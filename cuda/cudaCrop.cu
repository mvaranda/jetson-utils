#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

#include "cudaCrop.h"



// gpuCrop
template <typename T>
__global__ void gpuCrop( T* input, T* output, int offsetX, int offsetY, 
					int inWidth, int outWidth, int outHeight )
{
	const int out_x = blockIdx.x * blockDim.x + threadIdx.x;
	const int out_y = blockIdx.y * blockDim.y + threadIdx.y;

	if( out_x >= outWidth || out_y >= outHeight )
		return;

	const int in_x = out_x + offsetX;
	const int in_y = out_y + offsetY;

	output[out_y * outWidth + out_x] = input[in_y * inWidth + in_x];
}


// cudaCrop
hipError_t cudaCrop( float* input, float* output, const int4& roi, 
				  size_t inputWidth, size_t inputHeight )
{
	if( !input || !output )
		return hipErrorInvalidDevicePointer;

	if( inputWidth == 0 || inputHeight == 0 )
		return hipErrorInvalidValue;

	// get the ROI/output dimensions
	const int outputWidth = roi.z - roi.x;
	const int outputHeight = roi.w - roi.y;

	// validate the requested ROI
	if( outputWidth <= 0 || outputHeight <= 0 )
		return hipErrorInvalidValue;

	if( outputWidth > inputWidth || outputHeight > inputHeight )
		return hipErrorInvalidValue;

	if( roi.x < 0 || roi.y < 0 || roi.z < 0 || roi.w < 0 )
		return hipErrorInvalidValue;

	// launch kernel
	const dim3 blockDim(8, 8);
	const dim3 gridDim(iDivUp(outputWidth,blockDim.x), iDivUp(outputHeight,blockDim.y));

	gpuCrop<float><<<gridDim, blockDim>>>(input, output, roi.x, roi.y, inputWidth, outputWidth, outputHeight);

	return CUDA(hipGetLastError());
}


// cudaCropRGBA
hipError_t cudaCropRGBA( float4* input, float4* output, const int4& roi, 
					 size_t inputWidth, size_t inputHeight )
{
	if( !input || !output )
		return hipErrorInvalidDevicePointer;

	if( inputWidth == 0 || inputHeight == 0 )
		return hipErrorInvalidValue;

	// get the ROI/output dimensions
	const int outputWidth = roi.z - roi.x;
	const int outputHeight = roi.w - roi.y;

	// validate the requested ROI
	if( outputWidth <= 0 || outputHeight <= 0 )
		return hipErrorInvalidValue;

	if( outputWidth > inputWidth || outputHeight > inputHeight )
		return hipErrorInvalidValue;

	if( roi.x < 0 || roi.y < 0 || roi.z < 0 || roi.w < 0 )
		return hipErrorInvalidValue;

	// launch kernel
	const dim3 blockDim(8, 8);
	const dim3 gridDim(iDivUp(outputWidth,blockDim.x), iDivUp(outputHeight,blockDim.y));

	gpuCrop<float4><<<gridDim, blockDim>>>(input, output, roi.x, roi.y, inputWidth, outputWidth, outputHeight);

	return CUDA(hipGetLastError());
}





